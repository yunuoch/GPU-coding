#include "hip/hip_runtime.h"
#include "mex.h"
#include "gpu/mxGPUArray.h"

/*
 * Device code
 */


__global__ void MatMulKernel(double* A, double* B, double* C, dim3 dimsA, dim3 dimsB)//[1]  
{
	// Each thread computes one element of C  
	// by accumulating results into Cvalue  

	double Cvalue = 0;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	for (int e = 0; e < dimsA.y; ++e)
		Cvalue += A[row * dimsA.y + e] * B[e * dimsB.y + col];
	C[row * dimsB.y + col] = Cvalue;
}

void MatrixMultiplication_CUDA(const double* A, const double* B, double* C)
{
	dim3 dimsA(135, 135);// the size of matrix A which you have to modify  (4,3)
	dim3 dimsB(135, 3);// the size of matirx B which you have to modify   

					 //copy memory from host to devices  
	unsigned int size_A = dimsA.x * dimsA.y;
	unsigned int mem_size_A = sizeof(double) * size_A;
	double *d_A;
	hipMalloc(&d_A, mem_size_A);
	hipMemcpy(d_A, A, mem_size_A, hipMemcpyHostToDevice);
	double *d_B;
	unsigned int size_B = dimsB.x * dimsB.y;
	unsigned int mem_size_B = sizeof(double) * size_B;
	hipMalloc(&d_B, mem_size_B);
	hipMemcpy(d_B, B, mem_size_B, hipMemcpyHostToDevice);
	unsigned int mem_size_C = sizeof(double)* dimsA.x*dimsB.y;
	double *d_C;
	hipMalloc(&d_C, mem_size_C);

	//dimBlock represents the threads'size within block which you have to modify[2]  
	dim3 dimBlock(3, 2);
	dim3 dimGrid(dimsB.y / dimBlock.x, dimsA.x / dimBlock.y);//[3]  

	MatMulKernel << <dimGrid, dimBlock >> >(d_A, d_B, d_C, dimsA, dimsB);
	// Read C from device memory  
	hipMemcpy(C, d_C, mem_size_C,
		hipMemcpyDeviceToHost);
	// Free device memory  
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

}





/*
 * Host code
 */
void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{

    /*
     * Call the kernel using the CUDA runtime API. We are using a 1-d grid here,
     * and it would be possible for the number of elements to be too large for
     * the grid. For this example we are not guarding against this possibility.
     */
    //N = (int)(mxGPUGetNumberOfElements(A));
    //blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    //TimesTwo<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, N);

	int M0 = mxGetM(prhs[0]);    //得到arr0的行数
    int N0 = mxGetN(prhs[0]);    //得到arr0的列数
 //   double* pArr0 = (double*)mxGetPr(prhs[0]);//得到arr0的指针

    int M1 = mxGetM(prhs[1]);
    int N1 = mxGetN(prhs[1]);
//    double* pArr1 = (double*)mxGetPr(prhs[1]);


	
	
	double AA[] = { 6,2,3,
		8,3,5,
		7,2,4,
		8.3,2,5 };
	double BB[] = { 1,2,3,
		4,5,6,
		7,8,9 };
		//AA=[6,2,3;8,3,5;7,2,4;8.3,2,5 ];BB= [ 1,2,3;4,5,6;7,8,9 ];

	double* pArr0= (double*)mxGetPr(prhs[0]);
	double* pArr1= (double*)mxGetPr(prhs[1]);
	double*CC = new double[135*3];
//	MatrixMultiplication_CUDA(AA, BB, CC);
	MatrixMultiplication_CUDA(pArr0, pArr1, CC);
	
	plhs[0] = mxCreateDoubleMatrix(3, 135, mxREAL);
	double* pRe =(double*)mxGetPr(plhs[0]);
	for(int i=0;i<3*135;i++)
	{
	pRe[i]=CC[i];
	}
	
    /* Wrap the result up as a MATLAB gpuArray for return. */
    //plhs[0] = mxGPUCreateMxArrayOnGPU(B);

    /*
     * The mxGPUArray pointers are host-side structures that refer to device
     * data. These must be destroyed before leaving the MEX function.
     */

}
